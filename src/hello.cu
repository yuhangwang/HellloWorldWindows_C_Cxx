
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void 
helloFromGPU(void)
{	printf("hello world from GPU!\n");
}

int
main(int argc, char* argv[])
{	printf("Hello world from CPU!\n");
	helloFromGPU <<< 1, 10 >>>();
	hipDeviceReset();
	return 0;
}